#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
using namespace std;
#include <stdio.h>
#include "cutil_math.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

hipArray* d_transferFuncArray;
hipTextureObject_t transferTexObject; // Transfer texture Object

__constant__ unsigned int c_VF_data_scale = 256;
__constant__ unsigned int c_tex_width = 512;
__constant__ unsigned int c_tex_height = 512;
unsigned int h_VF_data_scale = 256;
unsigned int h_tex_width = 512;
unsigned int h_tex_height = 512;

extern "C" void checkCudaError(const char* msg)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA error: %s: %s. \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

typedef struct
{
	float4 m[3];
} float3x4;

__device__ float3 mul(const float3x4& M, const float3& v)
{
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}

__device__ float4 mul(const float3x4& M, const float4& v)
{
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = 1.0f;
	return r;
}

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

extern "C" void copyInvViewMatrix(float* invViewMatrix, size_t sizeofMatrix)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix);
	checkCudaError("Constant memcpy failed!");
}

struct Ray
{
	float3 origin;
	float3 dir;
};

// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm
__device__ bool intersectAABB(Ray ray, float3 boxMin, float3 boxMax, float* tNear, float* tFar)
{
	float3 invR = make_float3(1.0f) / ray.dir;
	float3 tBottom = invR * (boxMin - ray.origin);
	float3 tTop = invR * (boxMax - ray.origin);

	float3 tMin = fminf(tTop, tBottom);
	float3 tMax = fmaxf(tTop, tBottom);

	float largest_tMin = fmaxf(fmaxf(tMin.x, tMin.y), fmaxf(tMin.x, tMin.z));
	float smallest_tMax = fminf(fminf(tMax.x, tMax.y), fminf(tMax.x, tMax.z));

	*tNear = largest_tMin;
	*tFar = smallest_tMax;

	return smallest_tMax > largest_tMin;
}

extern "C" void createTransferTexture()
{
	// create transfer function texture
	float4 transferFunc[] =
	{
		{  0.0, 0.0, 0.0, 0.0, },
		{  1.0, 0.0, 0.0, 1.0, },
		{  1.0, 0.5, 0.0, 1.0, },
		{  1.0, 1.0, 0.0, 1.0, },
		{  0.0, 1.0, 0.0, 1.0, },
		{  0.0, 1.0, 1.0, 1.0, },
		{  0.0, 0.0, 1.0, 1.0, },
		{  1.0, 0.0, 1.0, 1.0, },
		{  0.0, 0.0, 0.0, 0.0, },
	};

	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	hipArray* d_transferFuncArray;
	hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1);
	checkCudaError("Cuda malloc transfer texture failed!");
	hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice);
	checkCudaError("Cuda memcpy transfer texture failed!");

	hipResourceDesc            texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_transferFuncArray;

	hipTextureDesc             texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = true; // access with normalized texture coordinates
	texDescr.filterMode = hipFilterModeLinear;

	texDescr.addressMode[0] = hipAddressModeClamp; // wrap texture coordinates

	texDescr.readMode = hipReadModeElementType;

	hipCreateTextureObject(&transferTexObject, &texRes, &texDescr, NULL);
	checkCudaError("Cuda create transfer texture failed!");
}

__device__ float3 Clamp_01(float3 p)
{
	float3 result;
	result.x = min(max(0.0f, p.x), 1.0f);
	result.y = min(max(0.0f, p.y), 1.0f);
	result.z = min(max(0.0f, p.z), 1.0f);

	return result;
}

__device__ unsigned int Index_xyz(int x, int y, int z, int N)
{
	return x * N * N + y * N + z;
}

__device__ unsigned int Index_uvw(float u, float v, float w, int N)
{
	unsigned int x = floor(u * N);
	unsigned int y = floor(v * N);
	unsigned int z = floor(w * N);

	x = min(max(0, x), N - 1);
	y = min(max(0, y), N - 1);
	z = min(max(0, z), N - 1);

	return x * N * N + y * N + z;
}

__global__ void fill_volume(float4* VF)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x >= c_VF_data_scale || y >= c_VF_data_scale || z >= c_VF_data_scale) return;

	unsigned int index = Index_xyz(x, y, z, c_VF_data_scale);
	
	// (0, 1)
	float u = (float)x / c_VF_data_scale;
	float v = (float)y / c_VF_data_scale;
	float w = (float)z / c_VF_data_scale;

	// (-1, 1)
	float u0 = u * 2.0f - 1.0f;
	float v0 = v * 2.0f - 1.0f;
	float w0 = w * 2.0f - 1.0f;

	// velocity
	VF[index].x = 0.0f;
	VF[index].y = 0.0f;
	VF[index].z = 0.0f;

	// Power
	//if (length(make_float3(u0, v0, w0) - make_float3(0.0f)) < 0.25f)
	if (u > 0.25f && u < 0.75f && v > 0.25f && v < 0.75f && w > 0.25f && w < 0.75f)
	{
		VF[index].w = 1.0f;
	}
		
}

extern "C" void launch_init_VF_kernel(float4* VF)
{
	dim3 block(8, 8, 8);
	dim3 grid(h_VF_data_scale / block.x, h_VF_data_scale / block.y, h_VF_data_scale / block.z);

	fill_volume << <grid, block >> > (VF);

	checkCudaError("Init VF kernel failed!");

	hipDeviceSynchronize();
}

__global__ void calculte_gradient(float4* VF, float3* gradient)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x >= c_VF_data_scale || y >= c_VF_data_scale || z >= c_VF_data_scale) return;

	unsigned int index = Index_xyz(x, y, z, c_VF_data_scale);

	// Gx of VF[index].w
	if (x == 0) gradient[index].x = VF[Index_xyz(x + 1, y, z, c_VF_data_scale)].w - VF[index].w;
	else if (x == c_VF_data_scale - 1) gradient[index].x = VF[index].w - VF[Index_xyz(x - 1, y, z, c_VF_data_scale)].w;
	else gradient[index].x = 0.5f * (VF[Index_xyz(x + 1, y, z, c_VF_data_scale)].w - VF[Index_xyz(x - 1, y, z, c_VF_data_scale)].w);
	// Gy of VF[index].w
	if (y == 0) gradient[index].y = VF[Index_xyz(x, y + 1, z, c_VF_data_scale)].w - VF[index].w;
	else if (y == c_VF_data_scale - 1) gradient[index].y = VF[index].w - VF[Index_xyz(x, y - 1, z, c_VF_data_scale)].w;
	else gradient[index].y = 0.5f * (VF[Index_xyz(x, y + 1, z, c_VF_data_scale)].w - VF[Index_xyz(x, y - 1, z, c_VF_data_scale)].w);
	// Gz of of VF[index].w
	if (z == 0) gradient[index].z = VF[Index_xyz(x, y, z + 1, c_VF_data_scale)].w - VF[index].w;
	else if (z == c_VF_data_scale - 1) gradient[index].z = VF[index].w - VF[Index_xyz(x, y, z - 1, c_VF_data_scale)].w;
	else gradient[index].z = 0.5f * (VF[Index_xyz(x, y, z + 1, c_VF_data_scale)].w - VF[Index_xyz(x, y, z - 1, c_VF_data_scale)].w);
}

extern "C" void launch_gradient_kernel(float4* VF, float3* gradient)
{
	dim3 block(8, 8, 8);
	dim3 grid(h_VF_data_scale / block.x, h_VF_data_scale / block.y, h_VF_data_scale / block.z);

	calculte_gradient << <grid, block >> > (VF, gradient);

	checkCudaError("Gradient kernel failed!");

	hipDeviceSynchronize();
}

__global__ void calculte_divergence(float3* gradient, float* divergence)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x >= c_VF_data_scale || y >= c_VF_data_scale || z >= c_VF_data_scale) return;

	unsigned int index = Index_xyz(x, y, z, c_VF_data_scale);

	float3 div = make_float3(0.0f);

	// Dx
	if (x == 0) div.x = gradient[Index_xyz(x + 1, y, z, c_VF_data_scale)].x - gradient[index].x;
	else if (x == c_VF_data_scale - 1) div.x = gradient[index].x - gradient[Index_xyz(x - 1, y, z, c_VF_data_scale)].x;
	else div.x = 0.5f * (gradient[Index_xyz(x + 1, y, z, c_VF_data_scale)].x - gradient[Index_xyz(x - 1, y, z, c_VF_data_scale)].x);
	// Dy
	if (y == 0) div.y = gradient[Index_xyz(x, y + 1, z, c_VF_data_scale)].y - gradient[index].y;
	else if (y == c_VF_data_scale - 1) div.y = gradient[index].y - gradient[Index_xyz(x, y - 1, z, c_VF_data_scale)].y;
	else div.y = 0.5f * (gradient[Index_xyz(x, y + 1, z, c_VF_data_scale)].y - gradient[Index_xyz(x, y - 1, z, c_VF_data_scale)].y);
	// Dz
	if (z == 0) div.z = gradient[Index_xyz(x, y, z + 1, c_VF_data_scale)].z - gradient[index].z;
	else if (z == c_VF_data_scale - 1) div.z = gradient[index].z - gradient[Index_xyz(x, y, z - 1, c_VF_data_scale)].z;
	else div.z = 0.5f * (gradient[Index_xyz(x, y, z + 1, c_VF_data_scale)].z - gradient[Index_xyz(x, y, z - 1, c_VF_data_scale)].z);

	divergence[index] = div.x + div.y + div.z;
}

extern "C" void launch_divergence_kernel(float3* gradient, float* divergence)
{
	dim3 block(8, 8, 8);
	dim3 grid(h_VF_data_scale / block.x, h_VF_data_scale / block.y, h_VF_data_scale / block.z);

	calculte_divergence << <grid, block >> > (gradient, divergence);

	checkCudaError("Divergence kernel failed!");

	hipDeviceSynchronize();
}

__global__ void updateVF(float4* pre_VF, float4* current_VF, float* divergence)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x >= c_VF_data_scale || y >= c_VF_data_scale || z >= c_VF_data_scale) return;

	unsigned int index = Index_xyz(x, y, z, c_VF_data_scale);

	current_VF[index].w = max(pre_VF[index].w + divergence[index], 0.0f);
}

extern "C" void launch_update_VF_kernel(float4* pre_VF, float4* current_VF, float* divergence)
{
	dim3 block(8, 8, 8);
	dim3 grid(h_VF_data_scale / block.x, h_VF_data_scale / block.y, h_VF_data_scale / block.z);

	updateVF << <grid, block >> > (pre_VF, current_VF, divergence);

	checkCudaError("Update VF kernel failed!");

	hipDeviceSynchronize();
}

__global__ void render(float4* VF, float3* gradient, float* divergence,
	float3* cuda_diffusion_result,
	float3* cuda_velocity_result,
	float density, float transferOffset, float transferScale, hipTextureObject_t transferTex)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= c_tex_width) || (j >= c_tex_height))
		return;
	int index = j * c_tex_width + i;

	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	float u = (i / (float)c_tex_width) * 2.0f - 1.0f;
	float v = (j / (float)c_tex_height) * 2.0f - 1.0f;

	// calculate camera ray in world space
	Ray ray;
	ray.origin = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	ray.dir = normalize(make_float3(u, -v, -2.0f));
	ray.dir = mul(c_invViewMatrix, ray.dir);

	// background color
	cuda_diffusion_result[index] = make_float3(0.0f);
	cuda_velocity_result[index] = make_float3(0.0f);

	// intersect with AABB
	float tNear, tFar;
	bool hit = intersectAABB(ray, boxMin, boxMax, &tNear, &tFar);

	if (!hit) return;
	if (tNear > tFar) return;

	// ray marching
	float4 diffusion_sum = make_float4(0.0f);
	float4 velocity_sum = make_float4(0.0f);

	// ray marching parameters
	const float opacityThreshold = 0.95f;

	//float3 dt_vec = 1.0f / (make_float3(c_VF_data_scale) * ray.dir);
	//float dt = min(dt_vec.x, min(dt_vec.y, dt_vec.z));
	float dt = 0.05f;
	float INF = 0.01f;
	
	float3 pos = ray.origin + tNear * ray.dir;
	for (float t = tNear; t < tFar; t += dt)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float3 rePos = pos * 0.5f + 0.5f;

		unsigned int sampleIndex = Index_uvw(rePos.x, rePos.y, rePos.z, c_VF_data_scale);
		float4 sample = VF[sampleIndex];

		// diffusion
		float4 diffusion_color = tex1D<float4>(transferTex, (sample.w - transferOffset) * transferScale);
		//float4 diffusion_color = make_float4(make_float3(sample.w), 1.0f);

		diffusion_color.w = sample.w;

		diffusion_sum.x +=  (1.0f - diffusion_sum.w) * diffusion_color.w * diffusion_color.x;
		diffusion_sum.y +=  (1.0f - diffusion_sum.w) * diffusion_color.w * diffusion_color.y;
		diffusion_sum.z +=  (1.0f - diffusion_sum.w) * diffusion_color.w * diffusion_color.z;

		diffusion_sum.w += (1.0f - diffusion_sum.w) * diffusion_color.w;

		// velocity
		float3 velocity = normalize(make_float3(sample.x, sample.y, sample.z)) * 2.0f - 1.0f;
		float4 velocity_color = make_float4(gradient[sampleIndex] * 0.5f + 0.5f, 1.0f);
		//float4 velocity_color = make_float4(make_float3(divergence[sampleIndex]) * 0.5f + 0.5f, 1.0f);

		velocity_color.w = sample.w;

		velocity_sum.x += (1.0f - velocity_sum.w) * velocity_color.w * velocity_color.x;
		velocity_sum.y += (1.0f - velocity_sum.w) * velocity_color.w * velocity_color.y;
		velocity_sum.z += (1.0f - velocity_sum.w) * velocity_color.w * velocity_color.z;

		velocity_sum.w += (1.0f - velocity_sum.w) * velocity_color.w;

		if (diffusion_sum.w > opacityThreshold && velocity_sum.w > opacityThreshold) break;


		pos += ray.dir * dt;
	}

	

	cuda_diffusion_result[index] = Clamp_01(make_float3(diffusion_sum.x, diffusion_sum.y, diffusion_sum.z));
	cuda_velocity_result[index] = Clamp_01(make_float3(velocity_sum.x, velocity_sum.y, velocity_sum.z));
	return;
}

extern "C" void launch_display_kernel(float4* VF, float3* gradient, float* divergence,
	float3* cuda_diffusion_result,
	float3* cuda_velocity_result,
	float density, float transferOffset, float transferScale)
{
	int tx = 8;
	int ty = 8;

	dim3 blocks(h_tex_width / tx + 1, h_tex_height / ty + 1);
	dim3 threads(tx, ty);

	render << <blocks, threads >> > (VF, gradient, divergence, cuda_diffusion_result, cuda_velocity_result,
		density, transferOffset, transferScale, transferTexObject);

	checkCudaError("Display kernel failed!");

	hipDeviceSynchronize();
}

__global__ void sampling_VF(float3* cuda_vbo_result, float4* VF, unsigned int vf_view_scale)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x >= vf_view_scale || y >= vf_view_scale || z >= vf_view_scale) return;

	float u = x / (float)vf_view_scale;
	float v = y / (float)vf_view_scale;
	float w = z / (float)vf_view_scale;

	// sample from VF
	unsigned int VF_index = Index_uvw(u, v, w, c_VF_data_scale);
	float3 velocity = make_float3(VF[VF_index].x, VF[VF_index].y, VF[VF_index].z);
	float3 color = normalize(velocity) * 2.0f - 1.0f;

	// offset
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	w = w * 2.0f - 1.0f;

	unsigned int vbo_index = Index_xyz(x, y, z, vf_view_scale);


	// fill VBO
	cuda_vbo_result[4 * vbo_index] = make_float3(u, v, w);// vert start cuda_vbo_result
	cuda_vbo_result[4 * vbo_index + 1] = color;
	cuda_vbo_result[4 * vbo_index + 2] = make_float3(u, v, w) + make_float3(0.0f, 0.01f, 0.0f) + velocity * 0.1f;	// vert end cuda_vbo_result
	cuda_vbo_result[4 * vbo_index + 3] = color;
}

extern "C" void launch_vbo_kernel(float3* cuda_vbo_result, float4* VF, unsigned int vf_view_scale)
{
	dim3 block(8, 8, 8);
	dim3 grid(h_VF_data_scale / block.x, h_VF_data_scale / block.y, h_VF_data_scale / block.z);

	sampling_VF << <grid, block >> > (cuda_vbo_result, VF, vf_view_scale);

	checkCudaError("VBO kernel failed!");

	hipDeviceSynchronize();
}

extern "C" void freeCudaTextureBuffers()
{
	hipDestroyTextureObject(transferTexObject);
	checkCudaError("Destroy texture object failed!");
	hipFreeArray(d_transferFuncArray);
	checkCudaError("Free volume array failed!");
}
