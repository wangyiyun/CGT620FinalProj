#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
using namespace std;
#include <stdio.h>
#include "cutil_math.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

typedef unsigned char VolumeType;
hipArray* d_volumeArray = 0;	// 3D texture Data
hipTextureObject_t volumeTexObject; // 3D texture Object
hipArray* d_transferFuncArray;
hipTextureObject_t transferTexObject; // Transfer texture Object

__constant__ float fadeOut = 0.5f;

extern "C" void checkCudaError(const char* msg)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA error: %s: %s. \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

typedef struct
{
	float4 m[3];
} float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

extern "C" void copyInvViewMatrix(float* invViewMatrix, size_t sizeofMatrix)
{
	hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix);
	checkCudaError("Constant memcpy failed!");
}

struct Ray
{
	float3 origin;
	float3 dir;
};

// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm
__device__ bool intersectAABB(Ray ray, float3 boxMin, float3 boxMax, float* tNear, float* tFar)
{
	float3 invR = make_float3(1.0f) / ray.dir;
	float3 tBottom = invR * (boxMin - ray.origin);
	float3 tTop = invR * (boxMax - ray.origin);

	float3 tMin = fminf(tTop, tBottom);
	float3 tMax = fmaxf(tTop, tBottom);

	float largest_tMin = fmaxf(fmaxf(tMin.x, tMin.y), fmaxf(tMin.x, tMin.z));
	float smallest_tMax = fminf(fminf(tMax.x, tMax.y), fminf(tMax.x, tMax.z));

	*tNear = largest_tMin;
	*tFar = smallest_tMax;

	return smallest_tMax > largest_tMin;
}

__device__ float3 mul(const float3x4& M, const float3& v)
{
	float3 r;
	r.x = dot(v, make_float3(M.m[0]));
	r.y = dot(v, make_float3(M.m[1]));
	r.z = dot(v, make_float3(M.m[2]));
	return r;
}

__device__ float4 mul(const float3x4& M, const float4& v)
{
	float4 r;
	r.x = dot(v, M.m[0]);
	r.y = dot(v, M.m[1]);
	r.z = dot(v, M.m[2]);
	r.w = 1.0f;
	return r;
}

__device__ unsigned int Index(int x, int y, int z, unsigned int N)
{
	return x * N * N + y * N + z;
}

extern "C" void copyVolumeTextures(void* h_volume, hipExtent volumeSize)
{
	// create 3D array
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
	hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize);
	checkCudaError("Cuda malloc 3D array failed!");

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width * sizeof(VolumeType), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_volumeArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);
	checkCudaError("Cuda memcpy 3D array failed!");

	hipResourceDesc            texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_volumeArray;

	hipTextureDesc             texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = true; // access with normalized texture coordinates
	texDescr.filterMode = hipFilterModeLinear; // linear interpolation

	texDescr.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;

	texDescr.readMode = hipReadModeNormalizedFloat;

	hipCreateTextureObject(&volumeTexObject, &texRes, &texDescr, NULL);
	checkCudaError("Cuda create volume texture object failed!");

	// create transfer function texture
	float4 transferFunc[] =
	{
		{  0.0, 0.0, 0.0, 0.0, },
		{  1.0, 0.0, 0.0, 1.0, },
		{  1.0, 0.5, 0.0, 1.0, },
		{  1.0, 1.0, 0.0, 1.0, },
		{  0.0, 1.0, 0.0, 1.0, },
		{  0.0, 1.0, 1.0, 1.0, },
		{  0.0, 0.0, 1.0, 1.0, },
		{  1.0, 0.0, 1.0, 1.0, },
		{  0.0, 0.0, 0.0, 0.0, },
	};

	hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
	hipArray* d_transferFuncArray;
	hipMallocArray(&d_transferFuncArray, &channelDesc2, sizeof(transferFunc) / sizeof(float4), 1);
	checkCudaError("Cuda malloc transfer texture failed!");
	hipMemcpyToArray(d_transferFuncArray, 0, 0, transferFunc, sizeof(transferFunc), hipMemcpyHostToDevice);
	checkCudaError("Cuda memcpy transfer texture failed!");

	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = d_transferFuncArray;

	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = true; // access with normalized texture coordinates
	texDescr.filterMode = hipFilterModeLinear;

	texDescr.addressMode[0] = hipAddressModeClamp; // wrap texture coordinates

	texDescr.readMode = hipReadModeElementType;

	hipCreateTextureObject(&transferTexObject, &texRes, &texDescr, NULL);
	checkCudaError("Cuda create transfer texture failed!");
}

__global__ void render_3D_texture(float3* result, unsigned int width, unsigned int height, hipTextureObject_t volumeTex,
	hipTextureObject_t transferTex,
	float density, float transferOffset, float transferScale, unsigned int N,
	float3* outputVF)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= width) || (j >= height))
		return;
	int index = j * width + i;

	const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
	const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

	float u = (i / (float)width) * 2.0f - 1.0f;
	float v = (j / (float)height) * 2.0f - 1.0f;

	// ray marching parameters
	const int maxSteps = 500;
	const float tStep = 0.01f;
	const float opacityThreshold = 0.95f;

	// calculate camera ray in world space
	Ray ray;
	ray.origin = make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
	ray.dir = normalize(make_float3(u, -v, -2.0f));
	ray.dir = mul(c_invViewMatrix, ray.dir);

	// background color
	result[index] = make_float3(0.0f);

	// intersect with AABB
	float tNear, tFar;
	bool hit = intersectAABB(ray, boxMin, boxMax, &tNear, &tFar);

	if (!hit) return;

	// ray marching
	float4 sum = make_float4(0.0f);
	float t = tNear;
	float3 pos = ray.origin + ray.dir * t;
	float3 step = ray.dir * tStep;

	for (int i = 0; i < maxSteps; i++)
	{
		// read from 3D texture
		// remap position to [0, 1] coordinates
		float3 rePos = pos * 0.5f + 0.5f;
		unsigned int x = floor(rePos.x) * N;
		unsigned int y = floor(rePos.y) * N;
		unsigned int z = floor(rePos.z) * N;
		unsigned int index = x * N * N + y * N + z;

		rePos -= outputVF[index] * 0.05f;

		float sample = tex3D<float>(volumeTex, rePos.x, rePos.y, rePos.z);
		float4 color = tex1D<float4>(transferTex, (sample - transferOffset) * transferScale);
		color.w *= density;
		
		color.x *= color.w;
		color.y *= color.w;
		color.z *= color.w;

		sum += color * (1.0f - sum.w);

		if (sum.w > opacityThreshold) break;

		t += tStep;
		if (t > tFar) break;

		pos += step;
	}
	
	result[index] = make_float3(sum.x, sum.y, sum.z);
	//if (c_invViewMatrix.m[0].x == 1.0f) result[index] = make_float3(0.0f, 0.0f, 1.0f);
	return;
}

extern "C" void launch_pbo_kernel(float3* result, unsigned int width, unsigned int height,
	float density, float transferOffset, float transferScale, unsigned int N, 
	float3* outputVF)
{
	int tx = 8;
	int ty = 8;

	dim3 blocks(width / tx + 1, height / ty + 1);
	dim3 threads(tx, ty);

	render_3D_texture << <blocks, threads >> > (result, width, height, volumeTexObject, transferTexObject, 
		density, transferOffset, transferScale, N, outputVF);

	checkCudaError("pbo kernel failed!");
}

__device__ unsigned int Index(int x, int y, int z, int N)
{
	return x * N * N + y * N + z;
}

__device__ float3 advect(float3* input, unsigned int N, unsigned int x, unsigned int y, unsigned int z, float time)
{
	
	unsigned int index = Index(x, y, z, N);
	int x0 = x - input[index].x;
	int y0 = y - input[index].y;
	int z0 = z - input[index].z;

	x0 = min(max(x0, 0), N - 1);
	y0 = min(max(y0, 0), N - 1);
	z0 = min(max(z0, 0), N - 1);

	return input[Index(x0, y0, z0, N)];
}

__global__ void update_vector_field(float3* pos, unsigned int N, unsigned int currentPickedIndex,
	float3 previewVect, hipTextureObject_t transferTex, float time, 
	float3* inputVF, float3* outputVF)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;

	if (x >= N || y >= N || z >= N) return;

	float u = x / (float)N;
	float v = y / (float)N;
	float w = z / (float)N;

	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;
	w = w * 2.0f - 1.0f;

	unsigned int index = x * N * N + y * N + z;
	//float len = length(inputVF[index]);
		
	//float4 color = tex1D<float4>(transferTex, len-0.1f);

	// Vector Field advect

	outputVF[index] = advect(inputVF, N, x, y, z, time);

	float3 dir = normalize(outputVF[index]);
	pos[4 * index] = make_float3(u, v, w);// vert start pos
	pos[4 * index + 1] = dir * 0.5f + 0.5f;
	pos[4 * index + 2] = make_float3(u, v, w) + make_float3(0.01f) + outputVF[index] * 0.1f;	// vert end pos
	pos[4 * index + 3] = dir * 0.5f + 0.5f;

	if (index == currentPickedIndex)
	{
		pos[4 * index + 2] = make_float3(u, v, w) + make_float3(0.01f) + previewVect * 0.1f;
		pos[4 * index + 1] = make_float3(1.0f, 1.0f, 0.0f);
		pos[4 * index + 3] = make_float3(1.0f, 1.0f, 0.0f);
	}
}

extern "C" void launch_vbo_kernel(float3* pos, unsigned int N, unsigned int currentPickedIndex, 
	float3 previewVect, float time, 
	float3* inputVF, float3* outputVF)
{
	dim3 block(4, 4, 4);
	dim3 grid(N / block.x, N / block.y, N / block.z);

	update_vector_field << <grid, block >> > (pos, N, currentPickedIndex, previewVect, transferTexObject, time,
		inputVF, outputVF);

	checkCudaError("vbo kernel failed!");

	hipDeviceSynchronize();
}

extern "C" void freeCudaBuffers()
{
	hipDestroyTextureObject(volumeTexObject);
	checkCudaError("Destroy texture object failed!");
	hipFreeArray(d_volumeArray);
	checkCudaError("Free volume array failed!");
	hipDestroyTextureObject(transferTexObject);
	checkCudaError("Destroy texture object failed!");
	hipFreeArray(d_transferFuncArray);
	checkCudaError("Free volume array failed!");
}
